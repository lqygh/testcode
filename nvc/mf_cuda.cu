#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

extern "C" {
#include "mf.h"
}

__global__ void frob(char* data, char* key, int datalen, uint16_t offset) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i < datalen) {
		data[i] ^= key[i+offset];
	}
}

extern "C"
void memfrob2(void* data, void* key, int datalen, uint16_t offset) {
	char* gpu_d;
	char* gpu_k;
	hipMalloc((void**)&gpu_d, datalen);
	hipMalloc((void**)&gpu_k, datalen);
	
	hipMemcpy(gpu_d, data, datalen, hipMemcpyHostToDevice);
	hipMemcpy(gpu_k, key, datalen, hipMemcpyHostToDevice);
	
	frob<<<(datalen + 255) / 256, 256>>>(gpu_d, gpu_k, datalen, offset);
	
	hipMemcpy(data, gpu_d, datalen, hipMemcpyDeviceToHost);
	
	hipFree(gpu_d);
	hipFree(gpu_k);
}
