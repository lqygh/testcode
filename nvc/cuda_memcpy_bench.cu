#include <stdio.h>
#include <stdlib.h>
#include <sys/timeb.h>
#include <hip/hip_runtime.h>

#define N 1500000000

int cudaCheck(hipError_t code) {
	if(code == hipSuccess) {
		//printf("cudaSuccess\n");
		return 0;
	} else {
		printf("cudaCheck(): %s\n", hipGetErrorString(hipGetLastError()));
		return -1;
	}
}

int main() {
	void* host = malloc(N);
	if(host == NULL) {
		perror("malloc()");
		return 1;
	}
	
	void* device = NULL;
	hipError_t ret = hipMalloc(&device, N);
	if(cudaCheck(ret) < 0) {
		return 1;
	}
	
	unsigned long int diff = 0;
	struct timeb t_begin = {0}, t_end = {0};
	
	ftime(&t_begin);
	hipMemcpy(host, device, N, hipMemcpyDeviceToHost);
	ftime(&t_end);
	diff = (t_end.time * 1000 + t_end.millitm) - (t_begin.time * 1000 + t_begin.millitm);
	printf("bytes transferred from device to host: %lu\n", (unsigned long int)N);
	printf("time elapsed: %lu ms\n", diff);
	printf("speed: %f MB/s\n", ((unsigned long int)N) / (1000.0 * diff));
	
	putchar('\n');
	
	ftime(&t_begin);
	hipMemcpy(device, host, N, hipMemcpyHostToDevice);
	ftime(&t_end);
	diff = (t_end.time * 1000 + t_end.millitm) - (t_begin.time * 1000 + t_begin.millitm);
	printf("bytes transferred from host to device: %lu\n", (unsigned long int)N);
	printf("time elapsed: %lu ms\n", diff);
	printf("speed: %f MB/s\n", ((unsigned long int)N) / (1000.0 * diff));
	
	free(host);
	hipFree(device);
	return 0;
}